#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include "common.h"
#include "bmp.h"
#include <stdlib.h>
#include <GL/freeglut.h>

#define DIM 512
#define blockSize 8

#define PI 3.1415926535897932f
#define centerX (DIM / 2)
#define centerY (DIM / 2)

float sourceColors[DIM * DIM];	// host memory for source image
float readBackPixels[DIM * DIM];	// host memory for swirled image

// Variable for memory allocation on the device
size_t size;

float* sourceDevPtr;			// device memory for source image
float* swirlDevPtr;				// device memory for swirled image

float userXCoord;						// user parameter for picture rotation on the X axis
float userYCoord;						// user parameter for picture rotation on the Y axis

__global__ void swirlKernel(float* sourcePtr, float* targetPtr, float userXCoord, float userYCoord)
{
	// TODO: Index berechnen	
	// blockDim(x, y, z) - number of threads in a block in each dimension
	// gridDim(x, y, z) - number of blocks in a grid in each dimension
	// blockIdx(x, y, z) - block index within a grid
	// threadIdx(x, y, z) - thread index within a block

	int tidX = threadIdx.x + blockIdx.x * blockDim.x;
	int tidY = threadIdx.y + blockIdx.y * blockDim.y;
	int index = tidX + tidY * blockDim.x * gridDim.x;

	// TODO: Den swirl invertieren.
	int relX = tidX - centerX;	// x position relative to center
	int relY = tidY - centerY;	// y position relative to center
	float originalAngle;		// angle before transformation, seen from a center based coordinate system

	if (relX != 0)
	{
		originalAngle = atan(((float)abs(relY) / (float)abs(relX)));

		if (relX > 0 && relY < 0)
			originalAngle = 2.0f * PI - originalAngle;
		else if (relX <= 0 && relY >= 0)
			originalAngle = PI - originalAngle;
		else if (relX <= 0 && relY < 0)
			originalAngle += PI;
	}
	else
	{
		// Take care of rare special case
		if (relY >= 0)
			originalAngle = 0.5f * PI;
		else
			originalAngle = 1.5f * PI;
	}

	// Calculate Rotation angle
	float r = sqrt((float)(relX * relX + relY * relY));
	float alpha = userXCoord * pow(r, userYCoord);

	float transformedAngle = originalAngle + alpha;

	// Transform in Pixel Coordinates
	int transX = (int)(floor(r * cos(transformedAngle) + 0.5f)) + centerX;
	int transY = (int)(floor(r * sin(transformedAngle) + 0.5f)) + centerY;

	// Clamping (handle edge cases of values less than or greater than the 
	// specified dimensions)
	if (transX < 0)
		transX = 0;
	if (transX >= DIM)
		transX = DIM - 1;
	if (transY < 0)
		transY = 0;
	if (transY >= DIM)
		transY = DIM - 1;

	// new Index
	int transIndex = transX + transY * blockDim.x * gridDim.x;

	targetPtr[transIndex] = sourcePtr[index];    // simple copy
}

void display(void)
{
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// TODO: Swirl Kernel aufrufen
	dim3 grid(blockSize * blockSize, blockSize * blockSize);
	dim3 block(blockSize, blockSize);
	swirlKernel<<<grid, block>>>(sourceDevPtr, swirlDevPtr, userXCoord, userYCoord);

	// TODO: Ergebnis zu host memory zuruecklesen.
	CUDA_SAFE_CALL(hipMemcpy(readBackPixels, swirlDevPtr, size, hipMemcpyDeviceToHost));

	// Ergebnis zeichnen (ja, jetzt gehts direkt wieder zur GPU zurueck...) 
	glDrawPixels(DIM, DIM, GL_LUMINANCE, GL_FLOAT, readBackPixels);

	glutSwapBuffers();
}

// clean up memory allocated on the GPU
void cleanup() {
	CUDA_SAFE_CALL(hipFree(sourceDevPtr));
	CUDA_SAFE_CALL(hipFree(swirlDevPtr));
}

void keyboard(int key, int x, int y)
{
	switch (key)
	{
		case GLUT_KEY_LEFT: userXCoord -= 0.01;
			if (userXCoord < -2) userXCoord = -2;
			printf("X-axis: %.2f , Y-axis: %.3f\r", userXCoord, userYCoord);
			break;
		case GLUT_KEY_RIGHT: userXCoord += 0.01;
			if (userXCoord > 2) userXCoord = 2;
			printf("X-axis: %.2f , Y-axis: %.3f\r", userXCoord, userYCoord);
			break;
		case GLUT_KEY_DOWN: userYCoord -= 0.001;
			if (userYCoord < 0) userYCoord = 0;
			printf("X-axis: %.2f , Y-axis: %.3f\r", userXCoord, userYCoord);
			break;
		case GLUT_KEY_UP: userYCoord += 0.001;
			if (userYCoord > 1) userYCoord = 1;
			printf("X-axis: %.2f , Y-axis: %.3f\r", userXCoord, userYCoord);
			break;
	}
}

int main(int argc, char **argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
	glutInitWindowSize(DIM, DIM);
	glutCreateWindow("Simple OpenGL CUDA");
	glutSpecialFunc(keyboard);
	glutIdleFunc(display);
	glutDisplayFunc(display);

	// load bitmap	
	Bitmap bmp = Bitmap("who-is-that.bmp");
	if (bmp.isValid())
	{
		for (int i = 0; i < DIM * DIM; i++)
		{
			sourceColors[i] = bmp.getR(i / DIM, i % DIM) / 255.0f;
		}
	}

	// Initalize parameters userXCoord and userYCoord
	userXCoord = 0.3;
	userYCoord = 0.7;

	// TODO: allocate memory at sourceDevPtr on the GPU and copy sourceColors into it.
	size = DIM * DIM * sizeof(float);
	CUDA_SAFE_CALL(hipMalloc((void**)&sourceDevPtr, size));
	CUDA_SAFE_CALL(hipMemcpy(sourceDevPtr, sourceColors, size, hipMemcpyHostToDevice));

	// TODO: allocate memory at swirlDevPtr for the unswirled image.
	CUDA_SAFE_CALL(hipMalloc((void**)&swirlDevPtr, size));

	glutMainLoop();

	cleanup();
}
