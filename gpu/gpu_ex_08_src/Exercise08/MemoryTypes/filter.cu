#include "hip/hip_runtime.h"

#include "common.h"
#include <stdlib.h>
#include <GL/freeglut.h>

#define DIM 512
#define blockSize 8
#define blurRadius 6
#define effectiveBlockSize (blockSize + 2 * blurRadius)

float sourceColors[DIM * DIM];
float readBackPixels[DIM * DIM];

texture<float, 2> blurDevTex;

float* sourceDevPtr;
float* transDevPtr;
float* blurDevPtr;

int timer = 0;

enum Mode
{
	NO_TRANSFORMATION,
	TRANSFORMATION,
	GLOBAL_MEMORY_BLUR,
	TEXTURE_MEMORY_BLUR,
	SHARED_MEMORY_BLUR
};
Mode mode = NO_TRANSFORMATION;

__global__ void transformation(float* sourcePtr, float* targetPtr, int timer) 
{
	int index = 0;
	int tidX = threadIdx.x + blockIdx.x + blockDim.x;		// thread x-coord in block
	int tidY = threadIdx.y + blockIdx.y + blockDim.y;		// thread y-coord in block
	index = (tidX + tidY) * blockDim.x * gridDim.x;		// block position in grid

	int transX = tidX;
	int transY = tidY;

	// define current x coord by getting the rest from
	// dividing with the current time 
	transX += timer % DIM;

	// Clamping: if x coord has values above DIM
	if (transX >= DIM)
	{
		// go to the next row of pixels by substracting DIM
		// and getting the new x coord
		transX -= DIM;
	}

	// define current y coord by getting the rest from
	// dividing with the current time 
	transY += timer % DIM;

	// Clamping: if y coord has values above DIM
	if (transY >= DIM)
	{
		// go to the next column of pixels by substracting DIM
		// and getting the new y coord
		transY -= DIM;
	}

	// get new index
	int transIndex = (transX + transY) * blockDim.x * gridDim.x;

	targetPtr[index] = sourcePtr[transIndex];    // simple copy
}

__global__ void globalMemoryBlur(float* sourcePtr, float* targetPtr)
{
	int tidX = threadIdx.x + blockIdx.x + blockDim.x;		// thread x-coord in block
	int tidY = threadIdx.y + blockIdx.y + blockDim.y;		// thread y-coord in block
	int index = (tidX + tidY) * blockDim.x * gridDim.x;		// block position in grid

	int filterWidth = blurRadius * 2 + 1;

	float median = 0.0f;

	int upperLeftFilterPosX = tidX - blurRadius;
	int upperLeftFilterPosY = tidY - blurRadius;

	for (int i = upperLeftFilterPosX; i < upperLeftFilterPosX + filterWidth; ++i)
	{
		for (int j = upperLeftFilterPosY; j < upperLeftFilterPosY + filterWidth; ++j)
		{
			if (i < DIM && j < DIM && i >= 0 && j >= 0)
			{
				// calculate index for neighboring pixel
				int sampleIndex = i + j * blockDim.x * gridDim.x;
				// add neighboring pixel's color in region of the radius
				median += sourcePtr[sampleIndex];
			}
		}
	}

	// get mean value 
	median /= filterWidth * filterWidth;

	targetPtr[index] = median;
}

__global__ void textureMemoryBlur(float* targetPtr)
{
	int tidX = threadIdx.x + blockIdx.x + blockDim.x;		// thread x-coord in block
	int tidY = threadIdx.y + blockIdx.y + blockDim.y;		// thread y-coord in block
	int index = (tidX + tidY) * blockDim.x * gridDim.x;		// block position in grid

	int filterWidth = blurRadius * 2 + 1;

	float median = 0.0f;

	int upperLeftFilterPosX = tidX - blurRadius;
	int upperLeftFilterPosY = tidY - blurRadius;

	for (int i = upperLeftFilterPosX; i < upperLeftFilterPosX + filterWidth; ++i)
	{
		for (int j = upperLeftFilterPosY; j < upperLeftFilterPosY + filterWidth; ++j)
		{
			if (i < DIM && j < DIM && i >= 0 && j >= 0)
			{
				median += tex2D(blurDevTex, j, i);
			}
		}
	}

	// get mean value 
	median /= filterWidth * filterWidth;

	targetPtr[index] = median;
}

__global__ void sharedMemoryBlur(float *sourcePtr, float *targetPtr)
{
	// calculate the position in source Image
	// therefore use blockSize not BlockDim.x
	int positionInImageX = blockIdx.x * blockSize + threadIdx.x - blurRadius;
	int positionInImageY = blockIdx.y * blockSize + threadIdx.y - blurRadius;

	__shared__ float cache[effectiveBlockSize * effectiveBlockSize];

	// fill the with values from global memory
	int getterIndex = positionInImageX + positionInImageY * DIM;

	if (0 <= positionInImageX && positionInImageX < DIM && 0 <= positionInImageY && positionInImageY < DIM)
	{
		cache[threadIdx.x + threadIdx.y * effectiveBlockSize] = sourcePtr[getterIndex];
	}
	else
	{
		cache[threadIdx.x + threadIdx.y * effectiveBlockSize] = 0.0f;
	}

	// synchronise all threads
	__syncthreads();

	// let all kernels run which have enough neighbors for mean calculation
	int kernelSizeRightSide = effectiveBlockSize - blurRadius;
	if (threadIdx.x >= blurRadius && threadIdx.x < kernelSizeRightSide && threadIdx.y >= blurRadius && threadIdx.y < kernelSizeRightSide)
	{
		float median = 0;
		for (int i = -blurRadius; i <= blurRadius; i++)
		{
			for (int j = -blurRadius; j <= blurRadius; j++)
			{
				median += cache[(threadIdx.x + j) + (threadIdx.y + i) * effectiveBlockSize];
			}
		}
		int filterWidth = blurRadius * 2 + 1;
		median /= filterWidth*filterWidth;
		targetPtr[positionInImageX + positionInImageY * DIM] = median;
	}
}

void keyboard(unsigned char key, int x, int y)
{
	switch (key)
	{
		case '1': 
			mode = TRANSFORMATION;
			break;
		case '2':
			mode = GLOBAL_MEMORY_BLUR;
			break;
		case '3':
			mode = TEXTURE_MEMORY_BLUR;
			break;
		case '4':
			mode = SHARED_MEMORY_BLUR;
			break;
	}
}

void display(void)	
{
	glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// TODO: Transformationskernel auf sourceDevPtr anwenden
	dim3 grid(DIM / blockSize, DIM / blockSize);
	dim3 block(blockSize, blockSize);

	dim3 sharedGrid(DIM / blockSize, DIM / blockSize);
	dim3 sharedBlock(effectiveBlockSize, effectiveBlockSize);

	if (mode == TRANSFORMATION)
	{
		timer += 1;
	}

	// TODO: Zeitmessung starten (see hipEventCreate, hipEventRecord)
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);


	switch (mode)
	{
		case TRANSFORMATION: 
			transformation<<<grid, block>>>(sourceDevPtr, transDevPtr, timer); 
			break;
		case GLOBAL_MEMORY_BLUR: 
			globalMemoryBlur<<<grid, block>>>(transDevPtr, blurDevPtr); 
			break;
		case TEXTURE_MEMORY_BLUR: 
			textureMemoryBlur<<<grid, block>>>(blurDevPtr); 
			break;
		case SHARED_MEMORY_BLUR: 
			sharedMemoryBlur<<<sharedGrid, sharedBlock>>>(transDevPtr, blurDevPtr); 
			break;
	}

	// TODO: Zeitmessung stoppen und fps ausgeben (see hipEventSynchronize, hipEventElapsedTime, hipEventDestroy)
	float elapsedTime;
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	printf("Time to generate: %3.1f ms \r", elapsedTime);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	// Ergebnis zur CPU zuruecklesen
	switch (mode)
	{
		case NO_TRANSFORMATION:
			CUDA_SAFE_CALL(hipMemcpy(readBackPixels, 
				sourceDevPtr, 
				DIM * DIM * 4, 
				hipMemcpyDeviceToHost));
			break;
		case TRANSFORMATION:
			CUDA_SAFE_CALL(hipMemcpy(readBackPixels, 
				transDevPtr, 
				DIM * DIM * 4, 
				hipMemcpyDeviceToHost));
			break;
		default: // bei Blur fuer restliche 3 Modi gleich
			CUDA_SAFE_CALL(hipMemcpy(readBackPixels, 
				blurDevPtr, 
				DIM * DIM * 4, 
				hipMemcpyDeviceToHost));
			break;
	}

	// Ergebnis zeichnen (ja, jetzt gehts direkt wieder zur GPU zurueck...) 
	glDrawPixels(DIM, DIM, GL_LUMINANCE, GL_FLOAT, readBackPixels);
	glutSwapBuffers();
}

// clean up memory allocated on the GPU
void cleanup() {
	CUDA_SAFE_CALL(hipFree(sourceDevPtr));
	// TODO: Aufr�umen zus�tzlich angelegter Ressourcen.
	CUDA_SAFE_CALL(hipUnbindTexture(blurDevTex));
	CUDA_SAFE_CALL(hipFree(transDevPtr));
	CUDA_SAFE_CALL(hipFree(blurDevPtr));
}

int main(int argc, char **argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
	glutInitWindowSize(DIM, DIM);
	glutCreateWindow("Memory Types");
	glutKeyboardFunc(keyboard);
	glutIdleFunc(display);
	glutDisplayFunc(display);

	// mit Schachbrettmuster f�llen
	for (int i = 0 ; i < DIM * DIM ; i++) {

		int x = (i % DIM) / (DIM / 8);
		int y = (i / DIM) / (DIM / 8);

		if ((x + y) % 2)
			sourceColors[i] = 1.0f;
		else
			sourceColors[i] = 0.0f;
	}

	// alloc memory on the GPU
	CUDA_SAFE_CALL(hipMalloc((void**)&sourceDevPtr, DIM * DIM * 4));
	CUDA_SAFE_CALL(hipMemcpy(sourceDevPtr, sourceColors, DIM * DIM * 4, hipMemcpyHostToDevice));

	// TODO: Weiteren Speicher auf der GPU f�r das Bild nach der Transformation und nach dem Blur allokieren.
	CUDA_SAFE_CALL(hipMalloc((void**)&transDevPtr, DIM * DIM * 4));
	CUDA_SAFE_CALL(hipMalloc((void**)&blurDevPtr, DIM * DIM * 4));

	// TODO: Binding des Speichers des Bildes an eine Textur mittels hipBindTexture.
	hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
	CUDA_SAFE_CALL(hipBindTexture2D(nullptr, blurDevTex, transDevPtr, desc, DIM, DIM, DIM * 4));

	glutMainLoop();

	cleanup();
}
