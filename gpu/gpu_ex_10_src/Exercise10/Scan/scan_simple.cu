#include "hip/hip_runtime.h"

#include <stdio.h>
#include "hip/hip_runtime.h"
#include <GL/glut.h>

#define N 512

GLfloat viewPosition[4] = {0.0f, 5.0f, 10.0f, 1.0f};  
GLfloat viewDirection[4] = {-0.0f, -5.0f, -10.0f, 0.0f};  
GLfloat viewAngle = 45.0f;
GLfloat viewNear = 4.5f;
GLfloat viewFar = 25.0f;

GLfloat xRotationAngle = 0.0f;
GLfloat yRotationAngle = 0.0f;

GLfloat xRotationSpeed = 3.0f;
GLfloat yRotationSpeed = 4.5f;

GLfloat depthPixels[N*N];
GLfloat colorPixels[N*N];
GLfloat filteredPixels[N*N];

float focusDepth = 0.5f;
float sizeScale = 20.0f;

float *devColorPixelsSrc, *devColorPixelsDst, *devDepthPixels, *devSAT;

void drawGround()
{
	GLfloat grey[3] = {0.8f, 0.8f, 0.8f};
    
	glNormal3f(0, 1, 0);
	glMaterialfv(GL_FRONT, GL_AMBIENT, grey);
	glMaterialfv(GL_FRONT, GL_DIFFUSE, grey);
    glBegin(GL_QUADS);
    glVertex3f(-10, 0, 10);
    glVertex3f( 10, 0, 10);
    glVertex3f( 10, 0, -10);
    glVertex3f(-10, 0, -10);
    glEnd();
}


void drawScene()
{
    GLfloat diffuse1[4]      = {0.5f, 0.5f, 0.5f, 1.0f};
	GLfloat lightAmbient[4]  = {0.0f, 0.0f, 0.0f, 1.0f};  
	GLfloat lightDiffuse[4]  = {0.2f, 0.2f, 0.2f, 1.0f};  
	GLfloat lightPosition[4] = {0.5f, 10.5f, 6.0f, 1.0f};  

	glLightfv(GL_LIGHT0, GL_AMBIENT, lightAmbient);
	glLightfv(GL_LIGHT0, GL_POSITION, lightPosition);
	glLightfv(GL_LIGHT0, GL_DIFFUSE, lightDiffuse);

    glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT, diffuse1);    
    glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, diffuse1);    
    glPushMatrix();
    glTranslatef(0.0f, 1.0f, 0.0f);
	glRotatef(-yRotationAngle/3.0f, 0.0f, 1.0f, 0.0f);
    glutSolidTeapot(1.0f);
    glPopMatrix();

	glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT, diffuse1);    
    glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, diffuse1);    
    glPushMatrix();
    glTranslatef(-1.0f, 1.0f, 3.0f);
	glRotatef(-yRotationAngle/3.0f, 0.0f, 1.0f, 0.0f);
    glutSolidTeapot(1.0f);
    glPopMatrix();

	glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT, diffuse1);    
    glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, diffuse1);    
    glPushMatrix();
    glTranslatef(1.0f, 1.0f, -3.0f);
	glRotatef(-yRotationAngle/3.0f, 0.0f, 1.0f, 0.0f);
    glutSolidTeapot(1.0f);
    glPopMatrix();

	glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT, diffuse1);    
    glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, diffuse1);    
    glPushMatrix();
    glTranslatef(-2.0f, 1.0f, 6.0f);
	glRotatef(-yRotationAngle/3.0f, 0.0f, 1.0f, 0.0f);
    glutSolidTeapot(1.0f);
    glPopMatrix();

	glMaterialfv(GL_FRONT_AND_BACK, GL_AMBIENT, diffuse1);    
    glMaterialfv(GL_FRONT_AND_BACK, GL_DIFFUSE, diffuse1);    
    glPushMatrix();
    glTranslatef(2.0f, 1.0f, -6.0f);
	glRotatef(-yRotationAngle/3.0f, 0.0f, 1.0f, 0.0f);
    glutSolidTeapot(1.0f);
    glPopMatrix();

	drawGround();
}



void initGL()
{
	glEnable(GL_DEPTH_TEST);
	glEnable(GL_LIGHTING);               // OpenGL Lichtquellen aktivieren
	glEnable(GL_LIGHT0);                 // Lichtquelle 0 anmachen 

	glMatrixMode(GL_PROJECTION);
	glLoadIdentity();
    gluPerspective(viewAngle, 1.0f, viewNear, viewFar);
	glMatrixMode(GL_MODELVIEW);
	glLoadIdentity();   
}

__device__ int clamp(int val, int minVal, int maxVal)
{
	// Ensure pixel coordinates are in the specified dimensions
	return min(max(val, minVal), maxVal);
}

__global__ void transpose(float* input, float* output)
{	
	int tidX = threadIdx.x + blockIdx.x * blockDim.x;
	int tidY = threadIdx.y + blockIdx.y * blockDim.y;

	// Calculate column index
	int columnIndex = tidY + tidX * blockDim.y * gridDim.y;	
	// Calculate row index
	int rowIndex = tidX + tidY * blockDim.x * gridDim.x;
 
	output[columnIndex] = input[rowIndex];
}

__global__ void sat_filter(float* dstImage,
							float *sat,
							float* srcDepth,
							float focusDepth, 
							float sizeScale,
							int n)
{
	/*int tidX = threadIdx.x + blockIdx.x * blockDim.x;
	int tidY = threadIdx.y + blockIdx.y * blockDim.y;*/
	int tidX = blockIdx.x;
	int tidY = blockIdx.y;
	int index = tidX + tidY * blockDim.y * gridDim.y;

	// TODO: Filtergr��e bestimmen
	// FilterSize = const * abs(zBuffer[x,y] �zFocus) + 1
	float filterSize = 1.0f + sizeScale * abs(srcDepth[index] - focusDepth);

	// Clamp coordinates on the X-axis
	int xLeft = clamp(tidX - filterSize / 2, 0, N - 1);
	int xRight = clamp(tidX + filterSize / 2, 0, N - 1);

	//Clamp coordinates on the Y-axis
	int yBottom = clamp(tidY - filterSize / 2, 0, N - 1);
	int yTop = clamp(tidY + filterSize / 2, 0, N - 1);

	// TODO: Anzahl der Pixel im Filterkern bestimmen	
	int pixelCount = (xRight - xLeft) * (yTop - yBottom);

	// TODO: SAT-Werte f�r die Eckpunkte des Filterkerns bestimmen.
	int satIndex = xRight + yTop * blockDim.y * gridDim.y;
	float a = sat[satIndex];

	satIndex = xLeft + yTop *blockDim.y * gridDim.y;
	float b = sat[satIndex];

	satIndex = xRight + yBottom * blockDim.y * gridDim.y;
	float c = sat[satIndex];

	satIndex = xLeft + yBottom * blockDim.y * gridDim.y;
	float d = sat[satIndex];

	// TODO: Mittelwert berechnen.
	float mean = (a - b - c + d) / ((float)pixelCount);
	dstImage[index] = mean;
}


__global__ void scan_naive(float *g_odata, float *g_idata, int n)
{
    // Dynamically allocated shared memory for scan kernels
    __shared__  float temp[2*N];

    int thid = threadIdx.x;
    int bid = blockIdx.x;

    int pout = 0;
    int pin = 1;

    // Cache the computational window in shared memory
    temp[pout * n + thid] = (thid > 0) ? g_idata[bid * N + thid - 1] : 0;

    for (int offset = 1; offset < n; offset *= 2)
    {
        pout = 1 - pout;
        pin  = 1 - pout;
        __syncthreads();

        temp[pout*n+thid] = temp[pin*n+thid];

        if (thid >= offset)
            temp[pout*n+thid] += temp[pin*n+thid - offset];
    }

    __syncthreads();

	g_odata[bid * N + thid] = temp[pout * n + thid];
}


void initCUDA()
{
	hipMalloc((void**)&devColorPixelsSrc, N * N * sizeof(float));
	hipMalloc((void**)&devColorPixelsDst, N * N * sizeof(float));
	hipMalloc((void**)&devDepthPixels, N * N * sizeof(float));
	hipMalloc((void**)&devSAT, N * N * sizeof(float));
}

void special(int key, int x, int y)
{
	switch (key) 
	{
		case GLUT_KEY_UP :
			focusDepth += 0.05f;
			 if (focusDepth > 1.0f) focusDepth = 1.0;
			break;
		case GLUT_KEY_DOWN :
			focusDepth -= 0.05f;
			 if (focusDepth < 0.0f) focusDepth = 0.0;
			break;
		case GLUT_KEY_LEFT :
			sizeScale -= 1.0f;
			if (sizeScale > 100.0f) sizeScale = 100.0;
			break;
		case GLUT_KEY_RIGHT :
			sizeScale += 1.0f;
			if (sizeScale < 1.0f) sizeScale = 1.0;
			break;
		case GLUT_KEY_PAGE_UP :
			viewFar += 1.0f;
			initGL();
			break;
		case GLUT_KEY_PAGE_DOWN :
			viewFar -= 1.0f;
			if (viewFar < viewNear) viewFar = viewNear;
			initGL();
			break;
	}
}

void display(void)								
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	// Szene rendern
	glLoadIdentity();
	gluLookAt(viewPosition[0], viewPosition[1], viewPosition[2], 
              viewDirection[0] - viewPosition[0], viewDirection[1] - viewPosition[1], viewDirection[2] - viewPosition[2], 
              0, 1, 0);    
	drawScene();	    

    // Tiefe und Farbe in den RAM streamen.
    glReadPixels( 0, 0, N, N, GL_DEPTH_COMPONENT, GL_FLOAT, depthPixels);	
    glReadPixels( 0, 0, N, N, GL_LUMINANCE, GL_FLOAT, colorPixels);	
	
    // Beide arrays in den Device-Memory kopieren.
	hipMemcpy(devColorPixelsSrc, colorPixels, N * N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(devDepthPixels, depthPixels, N * N * sizeof(float), hipMemcpyHostToDevice);

	dim3 blocks(N, N);
	hipMemset(devSAT, 0, N * N * sizeof(float));

	// TODO: Scan    
	scan_naive<<<N, N>>>(devColorPixelsDst, devColorPixelsSrc, N);
	// TODO: Transponieren    
	transpose<<<blocks, 1>>>(devColorPixelsDst, devSAT);
	// TODO: Scan  
	scan_naive<<<N, N>>>(devColorPixelsDst, devSAT, N);
	// TODO: Transponieren 
	transpose<<<blocks, 1>>>(devColorPixelsDst, devSAT);
	// TODO: SAT-Filter anwenden	
	sat_filter<<<blocks, 1>>>(devColorPixelsDst, devSAT, devDepthPixels, focusDepth, sizeScale, N);

	// Ergebnis in Host-Memory kopieren
	hipMemcpy(filteredPixels, devColorPixelsDst, N * N * 4, hipMemcpyDeviceToHost);

	// TODO: Beim #if aus der 0 eine 1 machen, damit das gefilterte Bild angezeigt wird!
#if 1
	// Mittelwert-Bild rendern
	glDrawPixels(N, N, GL_LUMINANCE, GL_FLOAT, filteredPixels);
#else
	// Durchreichen des Eingabebildes.
	glDrawPixels(N, N, GL_LUMINANCE, GL_FLOAT, colorPixels);
#endif

	xRotationAngle += xRotationSpeed;   // Rotationswinkel erhoehen
	yRotationAngle += yRotationSpeed;

	glutSwapBuffers();
}

int main(int argc, char **argv)
{
	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_DEPTH | GLUT_DOUBLE);
	glutInitWindowSize(N, N);
	glutCreateWindow("Simple CUDA SAT Depth of Field");

	glutDisplayFunc(display);
	glutIdleFunc(display);
	glutSpecialFunc(special);

	initGL();
	initCUDA();

	glutMainLoop();

	return 0;
}

